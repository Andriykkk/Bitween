#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include <mma.h>
#include <cstdio>

using namespace nvcuda;

// Profiling kernels to isolate performance bottlenecks
// Each kernel tests a specific part of the computation

// ============================================================================
// PART 1: Only load X into shared memory (no computation)
// ============================================================================
template<int WMMA_M, int WMMA_N, int WMMA_K, int BITS>
__global__ void profile_load_x_only(
    const half* __restrict__ x,
    half* __restrict__ out,
    int M, int N, int K
) {
    const int block_m = blockIdx.y;
    const int block_n = blockIdx.x;

    __shared__ half x_smem[WMMA_M][WMMA_K];

    float dummy = 0.0f;

    for (int k_start = 0; k_start < K; k_start += WMMA_K) {
        // LOAD X
        for (int idx = threadIdx.x; idx < WMMA_M * WMMA_K; idx += blockDim.x) {
            const int local_m = idx / WMMA_K;
            const int local_k = idx % WMMA_K;
            const int global_m = block_m * WMMA_M + local_m;
            const int global_k = k_start + local_k;

            if (global_m < M && global_k < K) {
                x_smem[local_m][local_k] = x[global_m * K + global_k];
                dummy += __half2float(x_smem[local_m][local_k]);
            }
        }
        __syncthreads();
    }

    // Prevent optimization
    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        out[0] = __float2half(dummy);
    }
}

// ============================================================================
// PART 2: Only load packed weights (no dequantization)
// ============================================================================
template<int WMMA_M, int WMMA_N, int WMMA_K, int BITS>
__global__ void profile_load_packed_only(
    const int* __restrict__ qweight,
    half* __restrict__ out,
    int M, int N, int K
) {
    constexpr int VALUES_PER_INT32 = 32 / BITS;
    const int packed_K = K / VALUES_PER_INT32;

    const int block_m = blockIdx.y;
    const int block_n = blockIdx.x;

    int dummy = 0;

    for (int k_start = 0; k_start < K; k_start += WMMA_K) {
        // LOAD PACKED WEIGHTS
        for (int idx = threadIdx.x; idx < WMMA_N * WMMA_K; idx += blockDim.x) {
            const int local_n = idx / WMMA_K;
            const int local_k = idx % WMMA_K;
            const int global_n = block_n * WMMA_N + local_n;
            const int global_k = k_start + local_k;

            if (global_n < N && global_k < K) {
                const int packed_idx = global_k / VALUES_PER_INT32;
                const int packed_val = qweight[global_n * packed_K + packed_idx];
                dummy += packed_val;
            }
        }
        __syncthreads();
    }

    // Prevent optimization
    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        out[0] = __float2half((float)dummy);
    }
}

// ============================================================================
// PART 3: Load packed + extract bits (no dequantization)
// ============================================================================
template<int WMMA_M, int WMMA_N, int WMMA_K, int BITS>
__global__ void profile_extract_bits_only(
    const int* __restrict__ qweight,
    half* __restrict__ out,
    int M, int N, int K
) {
    constexpr int VALUES_PER_INT32 = 32 / BITS;
    constexpr int QMASK = (1 << BITS) - 1;
    const int packed_K = K / VALUES_PER_INT32;

    const int block_m = blockIdx.y;
    const int block_n = blockIdx.x;

    int dummy = 0;

    for (int k_start = 0; k_start < K; k_start += WMMA_K) {
        for (int idx = threadIdx.x; idx < WMMA_N * WMMA_K; idx += blockDim.x) {
            const int local_n = idx / WMMA_K;
            const int local_k = idx % WMMA_K;
            const int global_n = block_n * WMMA_N + local_n;
            const int global_k = k_start + local_k;

            if (global_n < N && global_k < K) {
                const int packed_idx = global_k / VALUES_PER_INT32;
                const int bit_pos = global_k % VALUES_PER_INT32;

                const int packed_val = qweight[global_n * packed_K + packed_idx];
                // EXTRACT BITS
                const int q_val = (packed_val >> (bit_pos * BITS)) & QMASK;
                dummy += q_val;
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        out[0] = __float2half((float)dummy);
    }
}

// ============================================================================
// PART 4: Load scale/zero_point only
// ============================================================================
template<int WMMA_M, int WMMA_N, int WMMA_K, int BITS>
__global__ void profile_load_scale_zp_only(
    const half* __restrict__ scale,
    const half* __restrict__ zero_point,
    half* __restrict__ out,
    int M, int N, int K, int group_size
) {
    const int num_groups = (K + group_size - 1) / group_size;
    const int block_m = blockIdx.y;
    const int block_n = blockIdx.x;

    float dummy = 0.0f;

    for (int k_start = 0; k_start < K; k_start += WMMA_K) {
        for (int idx = threadIdx.x; idx < WMMA_N * WMMA_K; idx += blockDim.x) {
            const int local_n = idx / WMMA_K;
            const int local_k = idx % WMMA_K;
            const int global_n = block_n * WMMA_N + local_n;
            const int global_k = k_start + local_k;

            if (global_n < N && global_k < K) {
                // LOAD SCALE AND ZERO POINT
                const int group_idx = global_k / group_size;
                const half s = scale[global_n * num_groups + group_idx];
                const half z = zero_point[global_n * num_groups + group_idx];
                dummy += __half2float(s) + __half2float(z);
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        out[0] = __float2half(dummy);
    }
}

// ============================================================================
// PART 5: Full dequantization (no tensor core matmul)
// ============================================================================
template<int WMMA_M, int WMMA_N, int WMMA_K, int BITS>
__global__ void profile_dequantize_only(
    const int* __restrict__ qweight,
    const half* __restrict__ scale,
    const half* __restrict__ zero_point,
    half* __restrict__ out,
    int M, int N, int K, int group_size
) {
    constexpr int VALUES_PER_INT32 = 32 / BITS;
    constexpr int QMASK = (1 << BITS) - 1;
    const int packed_K = K / VALUES_PER_INT32;
    const int num_groups = (K + group_size - 1) / group_size;

    const int block_m = blockIdx.y;
    const int block_n = blockIdx.x;

    __shared__ half w_smem[WMMA_N][WMMA_K];

    float dummy = 0.0f;

    for (int k_start = 0; k_start < K; k_start += WMMA_K) {
        // FULL DEQUANTIZATION
        for (int idx = threadIdx.x; idx < WMMA_N * WMMA_K; idx += blockDim.x) {
            const int local_n = idx / WMMA_K;
            const int local_k = idx % WMMA_K;
            const int global_n = block_n * WMMA_N + local_n;
            const int global_k = k_start + local_k;

            if (global_n < N && global_k < K) {
                const int packed_idx = global_k / VALUES_PER_INT32;
                const int bit_pos = global_k % VALUES_PER_INT32;

                const int packed_val = qweight[global_n * packed_K + packed_idx];
                const int q_val = (packed_val >> (bit_pos * BITS)) & QMASK;

                const int group_idx = global_k / group_size;
                const half s = scale[global_n * num_groups + group_idx];
                const half z = zero_point[global_n * num_groups + group_idx];

                const half q_f = __int2half_rn(q_val);
                w_smem[local_n][local_k] = __hmul(s, __hsub(q_f, z));
                dummy += __half2float(w_smem[local_n][local_k]);
            } else {
                w_smem[local_n][local_k] = __float2half(0.0f);
            }
        }
        __syncthreads();
    }

    if (threadIdx.x == 0 && blockIdx.x == 0 && blockIdx.y == 0) {
        out[0] = __float2half(dummy);
    }
}

// ============================================================================
// PART 6: FP16 matmul only (no quantization)
// ============================================================================
template<int WMMA_M, int WMMA_N, int WMMA_K>
__global__ void profile_fp16_matmul_only(
    const half* __restrict__ x,
    const half* __restrict__ w,
    half* __restrict__ out,
    int M, int N, int K
) {
    const int block_m = blockIdx.y;
    const int block_n = blockIdx.x;

    __shared__ half x_smem[WMMA_M][WMMA_K];
    __shared__ half w_smem[WMMA_N][WMMA_K];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> acc_frag;

    wmma::fill_fragment(acc_frag, 0.0f);

    for (int k_start = 0; k_start < K; k_start += WMMA_K) {
        // Load X
        for (int idx = threadIdx.x; idx < WMMA_M * WMMA_K; idx += blockDim.x) {
            const int local_m = idx / WMMA_K;
            const int local_k = idx % WMMA_K;
            const int global_m = block_m * WMMA_M + local_m;
            const int global_k = k_start + local_k;

            if (global_m < M && global_k < K) {
                x_smem[local_m][local_k] = x[global_m * K + global_k];
            } else {
                x_smem[local_m][local_k] = __float2half(0.0f);
            }
        }

        // Load W
        for (int idx = threadIdx.x; idx < WMMA_N * WMMA_K; idx += blockDim.x) {
            const int local_n = idx / WMMA_K;
            const int local_k = idx % WMMA_K;
            const int global_n = block_n * WMMA_N + local_n;
            const int global_k = k_start + local_k;

            if (global_n < N && global_k < K) {
                w_smem[local_n][local_k] = w[global_n * K + global_k];
            } else {
                w_smem[local_n][local_k] = __float2half(0.0f);
            }
        }

        __syncthreads();

        // TENSOR CORE MATMUL
        wmma::load_matrix_sync(a_frag, &x_smem[0][0], WMMA_K);
        wmma::load_matrix_sync(b_frag, &w_smem[0][0], WMMA_K);
        wmma::mma_sync(acc_frag, a_frag, b_frag, acc_frag);

        __syncthreads();
    }

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, half> c_frag;
    for (int i = 0; i < c_frag.num_elements; ++i) {
        c_frag.x[i] = __float2half(acc_frag.x[i]);
    }

    const int out_m = block_m * WMMA_M;
    const int out_n = block_n * WMMA_N;

    if (out_m < M && out_n < N) {
        wmma::store_matrix_sync(&out[out_m * N + out_n], c_frag, N, wmma::mem_row_major);
    }
}

// ============================================================================
// Launcher functions
// ============================================================================

extern "C" void profile_load_x_cuda(const half* x, half* out, int M, int N, int K) {
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    dim3 grid((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block(32);
    profile_load_x_only<WMMA_M, WMMA_N, WMMA_K, 8><<<grid, block>>>(x, out, M, N, K);
}

extern "C" void profile_load_packed_cuda(const int* qweight, half* out, int M, int N, int K) {
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    dim3 grid((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block(32);
    profile_load_packed_only<WMMA_M, WMMA_N, WMMA_K, 8><<<grid, block>>>(qweight, out, M, N, K);
}

extern "C" void profile_extract_bits_cuda(const int* qweight, half* out, int M, int N, int K) {
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    dim3 grid((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block(32);
    profile_extract_bits_only<WMMA_M, WMMA_N, WMMA_K, 8><<<grid, block>>>(qweight, out, M, N, K);
}

extern "C" void profile_load_scale_zp_cuda(const half* scale, const half* zero_point, half* out,
                                            int M, int N, int K, int group_size) {
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    dim3 grid((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block(32);
    profile_load_scale_zp_only<WMMA_M, WMMA_N, WMMA_K, 8><<<grid, block>>>(
        scale, zero_point, out, M, N, K, group_size);
}

extern "C" void profile_dequantize_cuda(const int* qweight, const half* scale, const half* zero_point,
                                         half* out, int M, int N, int K, int group_size) {
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    dim3 grid((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block(32);
    profile_dequantize_only<WMMA_M, WMMA_N, WMMA_K, 8><<<grid, block>>>(
        qweight, scale, zero_point, out, M, N, K, group_size);
}

extern "C" void profile_fp16_matmul_cuda(const half* x, const half* w, half* out, int M, int N, int K) {
    constexpr int WMMA_M = 16, WMMA_N = 16, WMMA_K = 16;
    dim3 grid((N + WMMA_N - 1) / WMMA_N, (M + WMMA_M - 1) / WMMA_M);
    dim3 block(32);
    profile_fp16_matmul_only<WMMA_M, WMMA_N, WMMA_K><<<grid, block>>>(x, w, out, M, N, K);
}
